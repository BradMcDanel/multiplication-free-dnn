#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <vector>

#include "utils/checks.h"
#include "utils/cuda.cuh"
#include "quant_bn.h"

#include <ATen/cuda/HIPContext.h>

// Operations for reduce
template<typename T>
struct SumOp {
  __device__ SumOp(const T *t, int c, int s)
      : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ T operator()(int batch, int plane, int n) {
    return tensor[(batch * chn + plane) * sp + n];
  }
  const T *tensor;
  const int chn;
  const int sp;
};

template<typename T>
struct VarOp {
  __device__ VarOp(T m, const T *t, int c, int s)
      : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ T operator()(int batch, int plane, int n) {
    T val = tensor[(batch * chn + plane) * sp + n];
    return (val - mean) * (val - mean);
  }
  const T mean;
  const T *tensor;
  const int chn;
  const int sp;
};

template<typename T>
struct GradOp {
  __device__ GradOp(T _weight, T _bias, const T *_z, const T *_dz, int c, int s)
      : weight(_weight), bias(_bias), z(_z), dz(_dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<T> operator()(int batch, int plane, int n) {
    T _y = (z[(batch * chn + plane) * sp + n] - bias) / weight;
    T _dz = dz[(batch * chn + plane) * sp + n];
    return Pair<T>(_dz, _y * _dz);
  }
  const T weight;
  const T bias;
  const T *z;
  const T *dz;
  const int chn;
  const int sp;
};

/***********
 * mean_var
 ***********/

template<typename T>
__global__ void mean_var_kernel(const T *x, T *mean, T *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  T norm = T(1) / T(num * sp);

  T _mean = reduce<T, SumOp<T>>(SumOp<T>(x, chn, sp), plane, num, sp) * norm;
  __syncthreads();
  T _var = reduce<T, VarOp<T>>(VarOp<T>(_mean, x, chn, sp), plane, num, sp) * norm;

  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda(at::Tensor x) {
  CHECK_CUDA_INPUT(x);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Prepare output tensors
  auto mean = at::empty({chn}, x.options());
  auto var = at::empty({chn}, x.options());

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.type(), "mean_var_cuda", ([&] {
    mean_var_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        x.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        num, chn, sp);
  }));

  return {mean, var};
}

/**********
 * forward
 **********/

template<typename T>
__global__ void forward_kernel(T *x, const T *mean, const T *var, const T *weight, const T *bias,
                               bool affine, float eps, int num, int chn, int sp, 
                               int log_min_exp, int log_max_exp, float delta, float maxv) {
  int plane = blockIdx.x;

  float minv = -maxv;
  float clampv = minv;
  T _mean = mean[plane];
  T _var = var[plane];
  T _weight = affine ? abs(weight[plane]) + eps : T(1);
  T _bias = affine ? bias[plane] : T(0);
  T mul = rsqrt(_var + eps) * _weight;

  //log quantize mul
  T mul_exp = roundf(__log2f(abs(mul)));
  T mul_round = mul_exp < log_min_exp ? 0 :  __powf(2, fminf(mul_exp, log_max_exp));
  mul = (signbit(mul) ? -1 : 1) * mul_round;

  //linear quant _shift
  T _shift = -_mean*mul + _bias;
  _shift = (_shift < minv) ?
          clampv :
          fminf(fmaxf(delta*floor((_shift / delta) + 0.5), minv), maxv);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      T _x = x[(batch * chn + plane) * sp + n];
      T _y = _x * mul + _shift;
      x[(batch * chn + plane) * sp + n] = _y;
    }
  }
}

at::Tensor forward_cuda(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                        bool affine, float eps, int log_min_exp, int log_max_exp, float delta, float maxv) {
  CHECK_CUDA_INPUT(x);
  CHECK_CUDA_INPUT(mean);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.type(), "forward_cuda", ([&] {
    forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        x.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        affine, eps, num, chn, sp,
        log_min_exp, log_max_exp, delta, maxv);
  }));

  return x;
}

/***********
 * edz_eydz
 ***********/

template<typename T>
__global__ void edz_eydz_kernel(const T *z, const T *dz, const T *weight, const T *bias,
                                T *edz, T *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _weight = affine ? abs(weight[plane]) + eps : 1.f;
  T _bias = affine ? bias[plane] : 0.f;

  Pair<T> res = reduce<Pair<T>, GradOp<T>>(GradOp<T>(_weight, _bias, z, dz, chn, sp), plane, num, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                      bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty({chn}, z.options());
  auto eydz = at::empty({chn}, z.options());

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(z.type(), "edz_eydz_cuda", ([&] {
    edz_eydz_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        z.data<scalar_t>(),
        dz.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        edz.data<scalar_t>(),
        eydz.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return {edz, eydz};
}

/***********
 * backward
 ***********/

template<typename T>
__global__ void backward_kernel(const T *z, const T *dz, const T *var, const T *weight, const T *bias, const T *edz,
	                        const T *eydz, T *dx, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _weight = affine ? abs(weight[plane]) + eps : 1.f;
  T _bias = affine ? bias[plane] : 0.f;
  T _var = var[plane];
  T _edz = edz[plane];
  T _eydz = eydz[plane];

  T _mul = _weight * rsqrt(_var + eps);
  T count = T(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      T _dz = dz[(batch * chn + plane) * sp + n];
      T _y = (z[(batch * chn + plane) * sp + n] - _bias) / _weight;

      dx[(batch * chn + plane) * sp + n] = (_dz - _edz / count - _y * _eydz / count) * _mul;
    }
  }
}

at::Tensor backward_cuda(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                                      at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_CUDA_INPUT(z);
  CHECK_CUDA_INPUT(dz);
  CHECK_CUDA_INPUT(var);
  CHECK_CUDA_INPUT(weight);
  CHECK_CUDA_INPUT(bias);
  CHECK_CUDA_INPUT(edz);
  CHECK_CUDA_INPUT(eydz);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::zeros_like(z);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(z.type(), "backward_cuda", ([&] {
    backward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
        z.data<scalar_t>(),
        dz.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        edz.data<scalar_t>(),
        eydz.data<scalar_t>(),
        dx.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return dx;
}